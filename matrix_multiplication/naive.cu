#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < K) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[i * N + k] * B[k * K + j];
        }
        C[i * K + j] = sum;
    }
}

// Note: input_a, input_b, output_c are all device pointers to float32 arrays
extern "C" void solution(const float* input_a, const float* input_b, float* output_c, size_t m, size_t n, size_t k) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (m + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(input_a, input_b, output_c, m, k, n);
    hipDeviceSynchronize();
}
